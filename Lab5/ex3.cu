#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Error checking macro
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool
abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
        line);
        if (abort) exit(code);
    }
}

__global__ void reduceSum(int* input, int* output, int n)
{
    //Halving size of shared mem by reducing amount of mem req / thread.
    //It's done by performing the first reduction step while also writing to shared mem.
    //Ergo: Do first summation while filling up the shared mem => reducing overall mem usage.
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;
    
    //Doing the first reduction step while loading into shared memory
    sdata[tid] = input[start + tid] + input[start + blockDim.x + tid];
    
    //Unrolling the loop for stride values 512, 256, 128, 256, 512
    //Exec time using this technique = 3.38ms
    if (tid < blockDim.x / 32) sdata[tid] += sdata[tid + blockDim.x / 32];  // stride = 32
    __syncthreads();

    if (tid < blockDim.x / 64) sdata[tid] += sdata[tid + blockDim.x / 64];  // stride = 16
    __syncthreads();

    if (tid < blockDim.x / 128) sdata[tid] += sdata[tid + blockDim.x / 128];  // stride = 8
    __syncthreads();

    if (tid < blockDim.x / 256) sdata[tid] += sdata[tid + blockDim.x / 256];  // stride = 4
    __syncthreads();

    if (tid < blockDim.x / 512) sdata[tid] += sdata[tid + blockDim.x / 512];  // stride = 2
    __syncthreads();

    //Slower version with loop = 3.5ms
    /*for (unsigned int stride = blockDim.x; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (tid < stride)
            sdata[tid] += sdata[tid + stride];
    }

    __syncthreads();*/

    if (tid == 0)
        output[blockIdx.x] = sdata[0];
}


int main(void) {
    const int numElements = 1 << 24;
    const int threadsPerBlock = 512;
    const int blocksPerGrid = (numElements + threadsPerBlock * 2 - 1) /
    (threadsPerBlock * 2);
    const int smemSize = 2 * threadsPerBlock * sizeof(int);

    int *h_input = (int *)malloc(numElements * sizeof(int));
    int *h_output = (int *)malloc(blocksPerGrid * sizeof(int));

    // Initialize the host input vector
    for (int i = 0; i < numElements; ++i) {
    h_input[i] = rand() % 100;
    }

    int *d_input, *d_output;
    cudaCheckError(hipMalloc((void **)&d_input, numElements * sizeof(int)));
    cudaCheckError(hipMalloc((void **)&d_output, blocksPerGrid * sizeof(int)));

    cudaCheckError(hipMemcpy(d_input, h_input, numElements * sizeof(int),
    hipMemcpyHostToDevice));

    // Launch the reduction kernel
    reduceSum<<<blocksPerGrid, threadsPerBlock, smemSize>>>(d_input, d_output,
    numElements);
    cudaCheckError(hipGetLastError());

    cudaCheckError(hipMemcpy(h_output, d_output, blocksPerGrid * sizeof(int),
    hipMemcpyDeviceToHost));
    
    // Complete the reduction on the CPU
    int totalSum = 0;
    for (int i = 0; i < blocksPerGrid; ++i) {
        totalSum += h_output[i];
    }
    printf("Total Sum (GPU) = %d\n", totalSum);

    int totalSumCPU = 0;
    for (int i = 0; i < numElements; i++) {
        totalSumCPU += h_input[i];
    }

    printf("Total Sum (CPU) = %d\n", totalSumCPU);

    // Free device and host memory
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);

    return 0;
}