#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>
#include <jetson-utils/videoOutput.h>

__global__ void rgb2grayKernel (uchar4* image, uchar4* outPut, int w, int h)
{
    int total = w * h;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i <total ; i+=stride)
    {
        unsigned char gray = image[i].x * 0.299 + image[i].y * 0.587 + image[i].z * 0.114;
        outPut[i].x = gray;
        outPut[i].y = gray;
        outPut[i].z = gray;
    }
}


__global__ void calcHistogramKernel (uchar4 *image, int* hvector, int w, int h) {
    
    int total = w * h;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    //__shared__ int h_local[256];

    if (threadIdx.x < 256) {
        hvector[threadIdx.x] = 0;
    }
    __syncthreads();

    
    for (size_t i = index; i <total ; i+=stride) {
        unsigned char gray = image[i].x;
        atomicAdd(&hvector[gray], 1);
    }
   // __syncthreads();

    //update global histogram
    //if (threadIdx.x < 256) {
     //   atomicAdd(&hvector[threadIdx.x], h_local[threadIdx.x]);
    //}

}


int main( int argc, char** argv )
{
 // create input/output streams
 videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
 videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));
 videoOutput* output_2 = videoOutput::Create(argc,  argv, ARG_POSITION(1));

 uchar4* outPut = NULL;
 hipMalloc(&outPut, sizeof(uchar4) * 720 *1280);
 
 //host
 int hvector[256]= {0};
    

 //device 
 int* d_hvector = NULL;
 hipMalloc(&d_hvector, sizeof(int)*256);
 
 
 if ( !input )
    return 0;

 //capture/display loop
 while (true)
 {
     uchar4* image = NULL; // can be uchar3, uchar4, float3, float4
    int status = 0; // see videoSource::Status (OK, TIMEOUT, EOS, ERROR)
    if ( !input->Capture(&image, 1000, &status) ) // 1000ms timeout (default)
    {
        if (status == videoSource::TIMEOUT)
            continue;
            break; // EOS
    }

    if ( output != NULL )
    {
        hipMemset(d_hvector, 0, sizeof(int)*256);
        rgb2grayKernel<<<16,256>>> (image,outPut, input -> GetWidth(), input -> GetHeight());
        output->Render(outPut, input->GetWidth(), input->GetHeight());

        calcHistogramKernel<<<16,256>>>(outPut, d_hvector, input -> GetWidth(), input-> GetHeight());

        hipMemcpy(hvector, d_hvector, sizeof(int)*256, hipMemcpyDeviceToHost);
        //__syncthreads();

        int sum = 0;
        for (int i = 0; i < 256; i++)
            sum += hvector[i];

        //printf("Sum is %d", sum); //validate sum

        // Update status bar
        char str[256];
        sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
            input->GetHeight(), output->GetFrameRate());
        output->SetStatus(str);
        if (!output->IsStreaming()) // check if the user quit
        break;

        /*
        if( output_2 != NULL)
        {

            output_2-> Render(outPut, input -> GetWidth(), input-> GetHeight());

            // Update status bar
        char str[256];
        sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
            input->GetHeight(), output_2->GetFrameRate());
        output_2->SetStatus(str);
        if (!output_2->IsStreaming()) // check if the user quit
        break;
        }*/
    }
 }
hipFree(outPut);
} 