#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>
#include <jetson-utils/videoOutput.h>


__global__ void plotHistogramKernel(uchar4* image, int* histogram, int width, int height, int max_freq)
    {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    uchar4 white_pixel = make_uchar4(255, 255, 255, 255);
    //uchar4 black_pixel = make_uchar4(0, 0, 0, 255);
    float transparency_factor = 0.7;
    if (index < 256)
    {
    int freq = histogram[index] * 256 / max_freq;

    for (int i = 0; i < 256; i++)
    {
        int row = height - i - 1;
        int pixel_index= row * width +2 * index;
        if (i <= freq)
        {
            image[row * width + 2*index] = white_pixel;
            image[row * width + 2*index+1] = white_pixel;
        }
        else
        {
            image[row * width + 2*index].x *= transparency_factor;
            image[row * width + 2*index].y *= transparency_factor;
            image[row * width + 2*index].z *= transparency_factor;

            image[pixel_index +1 ].x *= transparency_factor;
            image[pixel_index +1 ].y *= transparency_factor;
            image[pixel_index +1 ].z *= transparency_factor;
        }
    }
    }
}

__global__ void rgb2grayKernel (uchar4* image, uchar4* outPut, int w, int h)
{
    int total = w * h;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (size_t i = index; i <total ; i+=stride)
    {
        unsigned char gray = image[i].x * 0.299 + image[i].y * 0.587 + image[i].z * 0.114;
        outPut[i].x = gray;
        outPut[i].y = gray;
        outPut[i].z = gray;
    }
}


__global__ void calcHistogramKernel (uchar4 *image, int* hvector, int w, int h) {
    
    int total = w * h;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    //__shared__ int h_local[256];

    if (threadIdx.x < 256) {
        hvector[threadIdx.x] = 0;
    }
    __syncthreads();

    
    for (size_t i = index; i <total ; i+=stride) {
        unsigned char gray = image[i].x;
        atomicAdd(&hvector[gray], 1);
    }
   // __syncthreads();

    //update global histogram
    //if (threadIdx.x < 256) {
     //   atomicAdd(&hvector[threadIdx.x], h_local[threadIdx.x]);
    //}

}


int main( int argc, char** argv )
{

 int max_freq= 20000;
 // create input/output streams
 videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
 videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));
 videoOutput* output_2 = videoOutput::Create(argc,  argv, ARG_POSITION(1));

 uchar4* outPut = NULL;
 hipMalloc(&outPut, sizeof(uchar4) * 720 *1280);
 
 //host
 int hvector[256]= {0};
    

 //device 
 int* d_hvector = NULL;
 hipMalloc(&d_hvector, sizeof(int)*256);
 
 
 if ( !input )
    return 0;

 //capture/display loop
 while (true)
 {
    uchar4* image = NULL; // can be uchar3, uchar4, float3, float4
    int status = 0; // see videoSource::Status (OK, TIMEOUT, EOS, ERROR)
    if ( !input->Capture(&image, 1000, &status) ) // 1000ms timeout (default)
    {
        if (status == videoSource::TIMEOUT)
            continue;
            break; // EOS
    }
    if ( output != NULL )
    {
        hipMemset(d_hvector, 0, sizeof(int)*256);
        rgb2grayKernel<<<16,256>>> (image,outPut, input -> GetWidth(), input -> GetHeight());
        calcHistogramKernel<<<16,256>>>(outPut, d_hvector, input -> GetWidth(), input-> GetHeight());
        plotHistogramKernel<<<1,256>>> (outPut, d_hvector, input -> GetWidth(), input -> GetHeight(), max_freq);

        hipMemcpy(hvector, d_hvector, sizeof(int)*256, hipMemcpyDeviceToHost);
        //__syncthreads();

        output->Render(outPut, input->GetWidth(), input->GetHeight());
        int sum = 0;

        for (int i = 0; i < 256; i++)
            sum += hvector[i];

        //printf("Sum is %d", sum); //Validate sum

        // Update status bar
        char str[256];
        sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
        input->GetHeight(), output->GetFrameRate());
        output->SetStatus(str);

        if (!output->IsStreaming()) // check if the user quit
        break;

        /*
        if( output_2 != NULL)
        {

            output_2-> Render(outPut, input -> GetWidth(), input-> GetHeight());

            // Update status bar
        char str[256];
        sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(),
            input->GetHeight(), output_2->GetFrameRate());
        output_2->SetStatus(str);
        if (!output_2->IsStreaming()) // check if the user quit
        break;
        }*/
    }
 }
hipFree(outPut);
hipFree(d_hvector);
} 